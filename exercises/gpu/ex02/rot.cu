
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <omp.h>
#include <math.h>

/***
 * coords structure
 ***/
typedef struct {
  float x;
  float y;
} coords;

/***
 * Print usage
 ***/
void
usage(char *argv[])
{
  fprintf(stderr, "usage: %s N\n", argv[0]);
  return;
}

/***
 * Allocate memory; print error if NULL is returned
 ***/
void *
ualloc(size_t size)
{
  void *ptr = malloc(size);
  if(ptr == NULL) {
    fprintf(stderr, "malloc() returned null; quitting...\n");
    exit(-2);
  }
  return ptr;
}

/***
 * Return seconds elapsed since t0, with t0 = 0 the epoch
 ***/
double
stop_watch(double t0)
{
  struct timeval t;
  gettimeofday(&t, NULL);
  return (double)t.tv_sec + (double)t.tv_usec/1e6 - t0;
}

/***
 * Read coords from binary file
 ***/
void
read_coords(coords *r, unsigned long int n, const char *fname)
{
  FILE *fp = fopen(fname, "r");
  for(int i=0; i<n; i++) {
    fread(&r[i].x, sizeof(float), 2, fp);
  }
  fclose(fp);
  return;
}

/***
 * Write coords to binary file
 ***/
void
write_coords(const char *fname, coords *r, unsigned long int n)
{
  FILE *fp = fopen(fname, "w");
  for(int i=0; i<n; i++) {
    fwrite(&r[i].x, sizeof(float), 2, fp);
  }
  fclose(fp);
  return;
}

/***
 * Do r' <- U*r + s on the CPU using OpenMP
 ***/
void
rotate(int n, coords *out, float theta, coords *r, coords *s)
{
  float ct = cos(theta);
  float st = sin(theta);
#pragma omp parallel for
  for(int i=0; i<n; i++) {
    out[i].x = ct*r[i].x - st*r[i].y + s[i].x;
    out[i].y = st*r[i].x + ct*r[i].y + s[i].y;
  }
  return;
}

int
main(int argc, char *argv[])
{
  /*
   * If number of arguments are not as expected, print usage and exit
   */
  if(argc != 2) {
    usage(argv);
    return 1;
  }

  unsigned long int n = atol(argv[1]);

  coords *r = (coords *)ualloc(sizeof(coords)*n);
  coords *s = (coords *)ualloc(sizeof(coords)*n);
  coords *v0 = (coords *)ualloc(sizeof(coords)*n);
  coords *v1 = (coords *)ualloc(sizeof(coords)*n);

  /*
   * Read from file
   */
  read_coords(r, n, "points.bin");
  read_coords(s, n, "shifts.bin");
    
  /*
   * The angle
   */
  float theta = (M_PI)*(10.0/180.0);
    
  /*
   * A: Run rotate(), return to v0, report performance
   */
  {
    double t0 = stop_watch(0);
    rotate(n, v0, theta, r, s);
    t0 = stop_watch(t0);

    double n_flop = 8;
    double n_io = 6*sizeof(float);
#pragma omp parallel
    {
#pragma omp single
      {
	int nthr = omp_get_num_threads();
	printf(" CPU: nthr = %4d   t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       nthr, t0, n_flop*n/1e9/t0, n_io*n/1e9/t0);
      }
    }
  }

  /*
   * B: Run rotate(), return to v1, report performance
   */
  {
    double t0 = stop_watch(0);
    rotate(n, v1, theta, r, s);
    t0 = stop_watch(t0);

    double n_flop = 8;
    double n_io = 6*sizeof(float);
#pragma omp parallel
    {
#pragma omp single
      {
	int nthr = omp_get_num_threads();
	printf(" CPU: nthr = %4d   t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       nthr, t0, n_flop*n/1e9/t0, n_io*n/1e9/t0);
      }
    }
  }

  /* Compare v1 and v0 */
  double diff = 0;
  double norm = 0;
  for(int i=0; i<n; i++) {
    float dx = v0[i].x-v1[i].x;
    float dy = v0[i].y-v1[i].y;
    diff += dx*dx + dy*dy;
    norm += v0[i].x*v0[i].x;
    norm += v0[i].y*v0[i].y;
  }
  printf(" Diff = %e\n", diff/norm);

  /*
   * Write to file
   */
  write_coords("points-rot.bin", v0, n);


  
  /*
   * Free arrays
   */
  free(r);
  free(s);
  free(v0);
  free(v1);
  return 0;
}
