
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <omp.h>

/***
 * Print usage
 ***/
void
usage(char *argv[])
{
  fprintf(stderr, "usage: %s M N\n", argv[0]);
  return;
}

/***
 * Allocate memory; print error if NULL is returned
 ***/
void *
ualloc(size_t size)
{
  void *ptr = malloc(size);
  if(ptr == NULL) {
    fprintf(stderr, "malloc() returned null; quitting...\n");
    exit(-2);
  }
  return ptr;
}

/***
 * Return a random number in [0, 1)
 ***/
double
urand(void)
{
  double x = (double)rand()/(double)RAND_MAX;
  return x;
}

/***
 * Return seconds elapsed since t0, with t0 = 0 the epoch
 ***/
double
stop_watch(double t0)
{
  struct timeval t;
  gettimeofday(&t, NULL);
  return (double)t.tv_sec + (double)t.tv_usec/1e6 - t0;
}

/***
 * Do y <- A*x on the CPU using OpenMP, y: m, A: mxn, x: n
 ***/
void
Ax(int m, int n, float *y, float *A, float *x)
{
#pragma omp parallel for
  for(int i=0; i<m; i++) {
    y[i] = 0.;
    for(int j=0; j<n; j++)
      y[i] += A[i*n + j]*x[j];
  }
  return;
}

int
main(int argc, char *argv[])
{
  /*
   * If number of arguments are not as expected, print usage and exit
   */
  if(argc != 3) {
    usage(argv);
    return 1;
  }

  unsigned long int m = atol(argv[1]);
  unsigned long int n = atol(argv[2]);

  float *x = (float *)ualloc(sizeof(float)*n);
  float *A = (float *)ualloc(sizeof(float)*n*m);
  float *y0 = (float *)ualloc(sizeof(float)*m);
  float *y1 = (float *)ualloc(sizeof(float)*m);

  /*
   * Initialize a and arrays
   */
  srand(2147483647);
  for(int i=0; i<n; i++) {
    x[i] = urand();
    for(int j=0; j<m; j++)
      A[i*m + j] = urand();
  }

  /*
   * A: Run Ax(), return to y0, report performance
   */
  {
    double t0 = stop_watch(0);
    Ax(m, n, y0, A, x);
    t0 = stop_watch(t0);

    double n_flop = 2*m*n;
    double n_io = sizeof(float)*(m*n + n + m);
#pragma omp parallel
    {
#pragma omp single
      {
	int nthr = omp_get_num_threads();
	printf(" CPU: nthr = %4d   t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       nthr, t0, n_flop/1e9/t0, n_io/1e9/t0);
      }
    }
  }

  /*
   * B: Run Ax(), return to y1, report performance
   */
  {
    double t0 = stop_watch(0);
    Ax(m, n, y1, A, x);
    t0 = stop_watch(t0);

    double n_flop = 2*m*n;
    double n_io = sizeof(float)*(m*n + n + m);
#pragma omp parallel
    {
#pragma omp single
      {
	int nthr = omp_get_num_threads();
	printf(" CPU: nthr = %4d   t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       nthr, t0, n_flop/1e9/t0, n_io/1e9/t0);
      }
    }
  }

  
  /* Compare y1 and y0 */
  double diff = 0;
  double norm = 0;
  for(int i=0; i<m; i++) {
    double d = y0[i]-y1[i];
    diff += d*d;
    norm += y0[i]*y0[i];
  }
  printf(" Diff = %e\n", diff/norm);
  /*
   * Don't need arrays anymore
   */
  free(x);
  free(A);
  free(y0);
  free(y1);
  return 0;
}
