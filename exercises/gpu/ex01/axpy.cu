
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <omp.h>

/***
 * Print usage
 ***/
void
usage(char *argv[])
{
  fprintf(stderr, "usage: %s N\n", argv[0]);
  return;
}

/***
 * Allocate memory; print error if NULL is returned
 ***/
void *
ualloc(size_t size)
{
  void *ptr = malloc(size);
  if(ptr == NULL) {
    fprintf(stderr, "malloc() returned null; quitting...\n");
    exit(-2);
  }
  return ptr;
}

/***
 * Return a random number in [0, 1)
 ***/
double
urand(void)
{
  double x = (double)rand()/(double)RAND_MAX;
  return x;
}

/***
 * Return seconds elapsed since t0, with t0 = 0 the epoch
 ***/
double
stop_watch(double t0)
{
  struct timeval t;
  gettimeofday(&t, NULL);
  return (double)t.tv_sec + (double)t.tv_usec/1e6 - t0;
}

/***
 * Do y <- a*x + y on the CPU using OpenMP
 ***/
void
axpy(int n, float a, float *x, float *y)
{
#pragma omp parallel for
  for(int i=0; i<n; i++)
    y[i] = a*x[i] + y[i];
  
  return;
}

int
main(int argc, char *argv[])
{
  /*
   * If number of arguments are not as expected, print usage and exit
   */
  if(argc != 2) {
    usage(argv);
    return 1;
  }

  unsigned long int n = atol(argv[1]);

  float *x0 = (float *)ualloc(sizeof(float)*n);
  float *x1 = (float *)ualloc(sizeof(float)*n);
  float *y0 = (float *)ualloc(sizeof(float)*n);
  float *y1 = (float *)ualloc(sizeof(float)*n);

  /*
   * Initialize a and arrays
   */
  srand(2147483647);
  float a = urand();
  for(int i=0; i<n; i++) {
    double rx = urand();
    x0[i] = rx;
    x1[i] = rx;

    double ry = urand();
    y0[i] = ry;
    y1[i] = ry;
  }

  /*
   * A: Run axpy(), return to y0, report performance
   */
  {
    double t0 = stop_watch(0);
    axpy(n, a, x0, y0);
    t0 = stop_watch(t0);

    double n_flop = 2;
    double n_io = 3*sizeof(float);
#pragma omp parallel
    {
#pragma omp single
      {
	int nthr = omp_get_num_threads();
	printf(" CPU: nthr = %4d   t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       nthr, t0, n_flop*n/1e9/t0, n_io*n/1e9/t0);
      }
    }
  }

  /*
   * B: Run axpy(), return to y1, report performance
   */
  {
    double t0 = stop_watch(0);
    axpy(n, a, x1, y1);
    t0 = stop_watch(t0);

    double n_flop = 2;
    double n_io = 3*sizeof(float);
#pragma omp parallel
    {
#pragma omp single
      {
	int nthr = omp_get_num_threads();
	printf(" CPU: nthr = %4d   t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       nthr, t0, n_flop*n/1e9/t0, n_io*n/1e9/t0);
      }
    }
  }

  /* Compare y1 and y0 */
  double diff = 0;
  double norm = 0;
  for(int i=0; i<n; i++) {
    float d = y0[i]-y1[i];
    diff += d*d;
    norm += y0[i]*y0[i];
  }
  printf(" Diff = %e\n", diff/norm);
  /*
   * Don't need arrays x and y anymore
   */
  free(x0);
  free(x1);
  free(y0);
  free(y1);
  return 0;
}
